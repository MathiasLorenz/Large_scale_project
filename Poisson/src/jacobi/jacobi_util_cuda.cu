#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "jacobi_util.h"
#include "jacobi_util_cuda.h"
#include "matrix_routines.h"
#include "cuda_routines.h"

// ============================================================================
// FUNCTIONS TO HANDLE THE INFORMATION STRUCTURE ON THE DEVICE

void copy_information_cuda(Information *information_cuda, Information *information)
{
	// Solution to use temporary structures was found at:
	// https://stackoverflow.com/questions/31133522/simple-operation-on-structure-in-cuda-segmentation-fault
	int size = information->size;

	// Allocate the temporary information structure
	Information Temp;

	// Simple Structure Elements
	Temp.size = information->size;
	Temp.rank = information->rank;
	Temp.global_Nx = information->global_Nx;
	Temp.global_Ny = information->global_Ny;
	Temp.global_Nz = information->global_Nz;

	// Allocate and copy the Arrays
	checkCudaErrors(hipMalloc( (void**) &Temp.loc_Nx, size*sizeof(int)));
	checkCudaErrors(hipMalloc( (void**) &Temp.loc_Ny, size*sizeof(int)));
	checkCudaErrors(hipMalloc( (void**) &Temp.loc_Nz, size*sizeof(int)));

	checkCudaErrors(hipMemcpyAsync(
		Temp.loc_Nx, 
		information->loc_Nx,
		information->size*sizeof(int), 
		hipMemcpyHostToDevice
	));
	checkCudaErrors(hipMemcpyAsync(
		Temp.loc_Ny, 
		information->loc_Ny,
		information->size*sizeof(int), 
		hipMemcpyHostToDevice
	));
	checkCudaErrors(hipMemcpyAsync(
		Temp.loc_Nz, 
		information->loc_Nz,
		information->size*sizeof(int), 
		hipMemcpyHostToDevice
	));

	Temp.maxit = information->maxit;
	Temp.tol = information->tol;
	Temp.use_tol = information->use_tol;
	Temp.norm_diff = information->norm_diff;

	checkCudaErrors(hipDeviceSynchronize());

	// Copy over the information structure
	checkCudaErrors(hipMemcpy(
		information_cuda, 
		&Temp,
		sizeof(Information), 
		hipMemcpyHostToDevice
	));
	
	checkCudaErrors(hipDeviceSynchronize());
}

void free_information_cuda(Information *information_cuda){
	free_information_arrays_cuda<<<1,1>>>(information_cuda);
	hipFree(information_cuda);
}

__global__ void free_information_arrays_cuda(Information *information_cuda)
{
	free(information_cuda->loc_Nx);
	free(information_cuda->loc_Ny);
	free(information_cuda->loc_Nz);
}

// ============================================================================
// CUDA VERSION OF THE ITERATIVE CORE

void jacobi_iteration_cuda(Information *information, Information *information_cuda,
	double *U_cuda, double *F_cuda, double *Unew_cuda)
{
	int K = information->global_Nx;
	int J = information->global_Ny;
	int I = information->global_Nz;

	dim3 BlockSize = dim3(32,32,32);
	dim3 BlockAmount = dim3( K/BlockSize.x + 1, J/BlockSize.y + 1, I/BlockSize.z + 1 );
	jacobi_iteration_kernel<<<BlockSize,BlockAmount>>>
		(information_cuda, U_cuda, F_cuda, Unew_cuda);
}

__global__ void jacobi_iteration_kernel(Information *information_cuda,
	double *U_cuda, double *F_cuda, double *Unew_cuda)
{
	// Determine where the thread is located
	int k = threadIdx.x + blockDim.x*blockIdx.x;
	int j = threadIdx.y + blockDim.y*blockIdx.y;
	int i = threadIdx.z + blockDim.z*blockIdx.z;

	// Read the needed data from the information structure
	int rank = information_cuda->rank;
	int Nx = information_cuda->global_Nx;
	int Ny = information_cuda->global_Ny;
	int Nz = information_cuda->global_Nz;
	int loc_Nx = information_cuda->loc_Nx[rank];
	int loc_Ny = information_cuda->loc_Ny[rank];
	int loc_Nz = information_cuda->loc_Nz[rank];

    int I, J, K;
	I = loc_Nz; J = loc_Ny; K = loc_Nx;

	if ( 
		( (i > 0) && (j > 0) && (k > 0)) 
		&& ((i < (I-1)) && (j < (J-1)) && (k < (K-1) )) ) 
	{
		// Setting up steps
		double hi = 2.0/(Nz-1.0);
		double hj = 2.0/(Ny-1.0);
		double hk = 2.0/(Nx-1.0);
		double stepi = hi*hi;
		double stepj = hj*hj;
		double stepk = hk*hk;
		double f3 = 1.0/3.0;
		double f6 = 1.0/6.0;

		// Compute new value
		// Save i, j, k index once
		int ijk = IND_3D(i, j, k, I, J, K);

		// Linear indexing with macro
		double ui = U_cuda[IND_3D(i - 1, j, k, I, J, K)] 
			+ U_cuda[IND_3D(i + 1, j, k, I, J, K)] 
			+ f3 * stepi * F_cuda[ijk];
		double uj = U_cuda[IND_3D(i, j - 1, k, I, J, K)] 
			+ U_cuda[IND_3D(i, j + 1, k, I, J, K)] 
			+ f3 * stepj * F_cuda[ijk];
		double uk = U_cuda[IND_3D(i, j, k - 1, I, J, K)] 
			+ U_cuda[IND_3D(i, j, k + 1, I, J, K)] 
			+ f3 * stepk * F_cuda[ijk];

		// Collect terms
		Unew_cuda[ijk] = f6 * (ui + uj + uk);
	}
}

// ============================================================================
// CUDA VERSION OF THE ITTERATIVE CORE
// This version handles interior and boundary separately

// Consider changing the BlockSize and BlockAmount for boundary call
void jacobi_iteration_cuda_separate(Information *information, Information *information_cuda,
	double *U_cuda, double *F_cuda, double *Unew_cuda, const char *ver)
{
	int K = information->global_Nx;
	int J = information->global_Ny;
	int I = information->global_Nz;

	// interior or boundary
	if (strcmp(ver, "i") == 0)
	{
		dim3 BlockSize = dim3(32, 32, 32);
		dim3 BlockAmount = dim3( K/BlockSize.x + 1, J/BlockSize.y + 1, I/BlockSize.z + 1 );
		jacobi_iteration_kernel_interior<<<BlockSize,BlockAmount>>>
				(information_cuda, U_cuda, F_cuda, Unew_cuda);
	}
	if (strcmp(ver, "b") == 0)   // boundary
	{
		dim3 BlockSize = dim3(32, 32, 1);
		dim3 BlockAmount = dim3( K/BlockSize.x + 1, J/BlockSize.y + 1, 1 );
		jacobi_iteration_kernel_boundary<<<BlockSize,BlockAmount>>>
				(information_cuda, U_cuda, F_cuda, Unew_cuda);
	}
}

// Kernel for interior points. Starts being used in mixed_3
__global__ void jacobi_iteration_kernel_interior(Information *information_cuda,
	double *U_cuda, double *F_cuda, double *Unew_cuda)
{
	// Determine where the thread is located
	int k = threadIdx.x + blockDim.x*blockIdx.x;
	int j = threadIdx.y + blockDim.y*blockIdx.y;
	int i = threadIdx.z + blockDim.z*blockIdx.z;

	// Read the needed data from the information structure
	int rank = information_cuda->rank;
	int Nx = information_cuda->global_Nx;
	int Ny = information_cuda->global_Ny;
	int Nz = information_cuda->global_Nz;
	int loc_Nx = information_cuda->loc_Nx[rank];
	int loc_Ny = information_cuda->loc_Ny[rank];
	int loc_Nz = information_cuda->loc_Nz[rank];

    int I, J, K;
	I = loc_Nz; J = loc_Ny; K = loc_Nx;

	// Setting up steps
	double hi = 2.0/(Nz-1.0);
	double hj = 2.0/(Ny-1.0);
	double hk = 2.0/(Nx-1.0);
	double stepi = hi*hi;
	double stepj = hj*hj;
	double stepk = hk*hk;
	double f3 = 1.0/3.0;
	double f6 = 1.0/6.0;

	// Interior case
	// Consider optimizing this as kernels for i \in {0, 1, K-2, K-1} are not used
	if ( 
		( (i > 1) && (j > 0) && (k > 0)) 
		&& ((i < (I-2)) && (j < (J-1)) && (k < (K-1) )) ) 
	{
		// Compute new value
		// Save i, j, k index once
		int ijk = IND_3D(i, j, k, I, J, K);

		// Linear indexing with macro
		double ui = U_cuda[IND_3D(i - 1, j, k, I, J, K)] 
			+ U_cuda[IND_3D(i + 1, j, k, I, J, K)] 
			+ f3 * stepi * F_cuda[ijk];
		double uj = U_cuda[IND_3D(i, j - 1, k, I, J, K)] 
			+ U_cuda[IND_3D(i, j + 1, k, I, J, K)] 
			+ f3 * stepj * F_cuda[ijk];
		double uk = U_cuda[IND_3D(i, j, k - 1, I, J, K)] 
			+ U_cuda[IND_3D(i, j, k + 1, I, J, K)] 
			+ f3 * stepk * F_cuda[ijk];

		// Collect terms
		Unew_cuda[ijk] = f6 * (ui + uj + uk);
	}
}

// Kernel for boundary points. Starts being used in mixed_3
__global__ void jacobi_iteration_kernel_boundary(Information *information_cuda,
	double *U_cuda, double *F_cuda, double *Unew_cuda)
{
	// Determine where the thread is located
	// Is super wrong for boundary now!
	int k = threadIdx.x + blockDim.x*blockIdx.x;
	int j = threadIdx.y + blockDim.y*blockIdx.y;

	// Read the needed data from the information structure
	int rank = information_cuda->rank;
	int Nx = information_cuda->global_Nx;
	int Ny = information_cuda->global_Ny;
	int Nz = information_cuda->global_Nz;
	int loc_Nx = information_cuda->loc_Nx[rank];
	int loc_Ny = information_cuda->loc_Ny[rank];
	int loc_Nz = information_cuda->loc_Nz[rank];

    int I, J, K;
	I = loc_Nz; J = loc_Ny; K = loc_Nx;

	// Setting up steps
	double hi = 2.0/(Nz-1.0);
	double hj = 2.0/(Ny-1.0);
	double hk = 2.0/(Nx-1.0);
	double stepi = hi*hi;
	double stepj = hj*hj;
	double stepk = hk*hk;
	double f3 = 1.0/3.0;
	double f6 = 1.0/6.0;

	// Boundary case
	if ( 
		( (j > 0) && (k > 0)) 
		&& ( (j < (J-1)) && (k < (K-1) )) ) 
	{
		// Compute new value
		// Do for first boundary (i = 1)
		int i = 1;
		int ijk = IND_3D(i, j, k, I, J, K);

		// Linear indexing with macro
		double ui = U_cuda[IND_3D(i - 1, j, k, I, J, K)] 
			+ U_cuda[IND_3D(i + 1, j, k, I, J, K)] 
			+ f3 * stepi * F_cuda[ijk];
		double uj = U_cuda[IND_3D(i, j - 1, k, I, J, K)] 
			+ U_cuda[IND_3D(i, j + 1, k, I, J, K)] 
			+ f3 * stepj * F_cuda[ijk];
		double uk = U_cuda[IND_3D(i, j, k - 1, I, J, K)] 
			+ U_cuda[IND_3D(i, j, k + 1, I, J, K)] 
			+ f3 * stepk * F_cuda[ijk];

		// Collect terms
		Unew_cuda[ijk] = f6 * (ui + uj + uk);


		// Do for last boundary (i = I - 1)
		i = I - 2;
		ijk = IND_3D(i, j, k, I, J, K);

		// Linear indexing with macro
		ui = U_cuda[IND_3D(i - 1, j, k, I, J, K)] 
			+ U_cuda[IND_3D(i + 1, j, k, I, J, K)] 
			+ f3 * stepi * F_cuda[ijk];
		uj = U_cuda[IND_3D(i, j - 1, k, I, J, K)] 
			+ U_cuda[IND_3D(i, j + 1, k, I, J, K)] 
			+ f3 * stepj * F_cuda[ijk];
		uk = U_cuda[IND_3D(i, j, k - 1, I, J, K)] 
			+ U_cuda[IND_3D(i, j, k + 1, I, J, K)] 
			+ f3 * stepk * F_cuda[ijk];

		// Collect terms
		Unew_cuda[ijk] = f6 * (ui + uj + uk);
	}
}
