#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "jacobi_util.h"
#include "jacobi_util_cuda.h"
#include "matrix_routines.h"
#include "cuda_routines.h"

// ============================================================================
// FUNCTIONS TO HANDLE THE INFORMATION STRUCTURE ON THE DEVICE

void copy_information_cuda(Information *information_cuda, Information *information)
{
	// Solution to use temporary structures was found at:
	// https://stackoverflow.com/questions/31133522/simple-operation-on-structure-in-cuda-segmentation-fault
	int size = information->size;

	// Allocate the temporary information structure
	Information Temp;

	// Simple Structure Elements
	Temp.size = information->size;
	Temp.rank = information->rank;
	Temp.global_Nx = information->global_Nx;
	Temp.global_Ny = information->global_Ny;
	Temp.global_Nz = information->global_Nz;

	// Allocate and copy the Arrays
	checkCudaErrors(hipMalloc( (void**) &Temp.loc_Nx, size*sizeof(int)));
	checkCudaErrors(hipMalloc( (void**) &Temp.loc_Ny, size*sizeof(int)));
	checkCudaErrors(hipMalloc( (void**) &Temp.loc_Nz, size*sizeof(int)));

	checkCudaErrors(hipMemcpyAsync(
		Temp.loc_Nx, 
		information->loc_Nx,
		information->size*sizeof(int), 
		hipMemcpyHostToDevice
	));
	checkCudaErrors(hipMemcpyAsync(
		Temp.loc_Ny, 
		information->loc_Ny,
		information->size*sizeof(int), 
		hipMemcpyHostToDevice
	));
	checkCudaErrors(hipMemcpyAsync(
		Temp.loc_Nz, 
		information->loc_Nz,
		information->size*sizeof(int), 
		hipMemcpyHostToDevice
	));
	
	Temp.maxit				= information->maxit;
	Temp.iter				= information->iter;
	Temp.tol				= information->tol;
	Temp.use_tol			= information->use_tol;
	Temp.local_frobenius	= information->local_frobenius;
	Temp.frobenius_error	= information->frobenius_error;

	checkCudaErrors(hipDeviceSynchronize());

	// Copy over the information structure
	checkCudaErrors(hipMemcpy(
		information_cuda, 
		&Temp,
		sizeof(Information), 
		hipMemcpyHostToDevice
	));
	
	checkCudaErrors(hipDeviceSynchronize());
}

void free_information_cuda(Information *information_cuda){
	free_information_arrays_cuda<<<1,1>>>(information_cuda);
	hipFree(information_cuda);
}

// As we these free's are launched in a kernel we de a normal free and
// NOT a hipFree.
__global__ void free_information_arrays_cuda(Information *information_cuda)
{
	free(information_cuda->loc_Nx);
	free(information_cuda->loc_Ny);
	free(information_cuda->loc_Nz);
}

// ============================================================================
// CUDA VERSION OF THE ITERATIVE CORE

void jacobi_iteration_cuda(Information *information, Information *information_cuda,
	double *U_cuda, double *F_cuda, double *Unew_cuda)
{
	int rank = information->rank;
	int K = information->loc_Nx[rank];
	int J = information->loc_Ny[rank];
	int I = information->loc_Nz[rank];

	// Blocks
	dim3 BlockSize = dim3(16,16,4);
	dim3 BlockAmount = dim3( K/BlockSize.x + 3, J/BlockSize.y + 3, I/BlockSize.z + 3 );

	jacobi_iteration_kernel<<<BlockAmount,BlockSize>>>
		(information_cuda, U_cuda, F_cuda, Unew_cuda);
}

__global__ void jacobi_iteration_kernel(Information *information_cuda,
	double *U_cuda, double *F_cuda, double *Unew_cuda)
{
	// Determine where the thread is located
	int k = threadIdx.x + blockDim.x*blockIdx.x;
	int j = threadIdx.y + blockDim.y*blockIdx.y;
	int i = threadIdx.z + blockDim.z*blockIdx.z;

	// Read the needed data from the information structure
	int rank = information_cuda->rank;
	int Nx = information_cuda->global_Nx;
	int Ny = information_cuda->global_Ny;
	int Nz = information_cuda->global_Nz;
	int loc_Nx = information_cuda->loc_Nx[rank];
	int loc_Ny = information_cuda->loc_Ny[rank];
	int loc_Nz = information_cuda->loc_Nz[rank];

	// For relative error stopping
	information_cuda->local_frobenius = 0.0;

    int I, J, K;
	I = loc_Nz; J = loc_Ny; K = loc_Nx;

	// Compute grid points
	if ( 
		( (i > 0) && (j > 0) && (k > 0)) 
		&& ((i < (I-1)) && (j < (J-1)) && (k < (K-1) )) ) 
	{
		// Setting up steps
		double hi = 2.0/(Nz-1.0);
		double hj = 2.0/(Ny-1.0);
		double hk = 2.0/(Nx-1.0);
		double stepi = hi*hi;
		double stepj = hj*hj;
		double stepk = hk*hk;
		double f3 = 1.0/3.0;
		double f6 = 1.0/6.0;

		// Compute new value
		// Save i, j, k index once
		int ijk = IND_3D(i, j, k, I, J, K);

		// Linear indexing with macro
		double ui = U_cuda[IND_3D(i - 1, j, k, I, J, K)] 
			+ U_cuda[IND_3D(i + 1, j, k, I, J, K)] 
			+ f3 * stepi * F_cuda[ijk];
		double uj = U_cuda[IND_3D(i, j - 1, k, I, J, K)] 
			+ U_cuda[IND_3D(i, j + 1, k, I, J, K)] 
			+ f3 * stepj * F_cuda[ijk];
		double uk = U_cuda[IND_3D(i, j, k - 1, I, J, K)] 
			+ U_cuda[IND_3D(i, j, k + 1, I, J, K)] 
			+ f3 * stepk * F_cuda[ijk];

		// Collect terms
		Unew_cuda[ijk] = f6 * (ui + uj + uk);
	}
}

// ============================================================================
// CUDA VERSION OF THE ITTERATIVE CORE
// This version handles interior and boundary separately

// Consider changing the BlockSize and BlockAmount for boundary call
void jacobi_iteration_cuda_separate(Information *information, Information *information_cuda,
	double *U_cuda, double *F_cuda, double *Unew_cuda, const char *ver)
{
	int rank = information->rank;
	int K = information->loc_Nx[rank];
	int J = information->loc_Ny[rank];
	int I = information->loc_Nz[rank];

	// interior or boundary
	if (strcmp(ver, "i") == 0)
	{
		dim3 BlockSize = dim3(16, 16, 4);
		dim3 BlockAmount = dim3( K/BlockSize.x + 3, J/BlockSize.y + 3, I/BlockSize.z + 3 );
		jacobi_iteration_kernel_interior<<<BlockAmount,BlockSize>>>
				(information_cuda, U_cuda, F_cuda, Unew_cuda);
	}
	if (strcmp(ver, "b") == 0)   // boundary
	{
		dim3 BlockSize = dim3(32, 16, 2);
		dim3 BlockAmount = dim3( K/BlockSize.x + 3, J/BlockSize.y + 3, 1 );
		jacobi_iteration_kernel_boundary<<<BlockAmount,BlockSize>>>
				(information_cuda, U_cuda, F_cuda, Unew_cuda);
	}
}
void jacobi_iteration_cuda_separate_stream(Information *information, Information *information_cuda,
	double *U_cuda, double *F_cuda, double *Unew_cuda, const char *ver,void *stream)
{
	hipStream_t *s;
	s = (hipStream_t*)stream;
	int rank = information->rank;
	int K = information->loc_Nx[rank];
	int J = information->loc_Ny[rank];
	int I = information->loc_Nz[rank];

	// interior or boundary
	if (strcmp(ver, "i") == 0)
	{
		dim3 BlockSize = dim3(16, 16, 4);
		dim3 BlockAmount = dim3( K/BlockSize.x + 3, J/BlockSize.y + 3, I/BlockSize.z + 3 );
		jacobi_iteration_kernel_interior<<<BlockAmount,BlockSize,0,*s>>>
				(information_cuda, U_cuda, F_cuda, Unew_cuda);
	}
	if (strcmp(ver, "b") == 0)   // boundary
	{
		dim3 BlockSize = dim3(32, 16, 2);
		dim3 BlockAmount = dim3( K/BlockSize.x + 3, J/BlockSize.y + 3, 1 );
		jacobi_iteration_kernel_boundary<<<BlockAmount,BlockSize,0,*s>>>
				(information_cuda, U_cuda, F_cuda, Unew_cuda);
	}
}


// Kernel for interior points. Starts being used in mixed_3
__global__ void jacobi_iteration_kernel_interior(Information *information_cuda,
	double *U_cuda, double *F_cuda, double *Unew_cuda)
{
	// Determine where the thread is located
	int k = threadIdx.x + blockDim.x*blockIdx.x;
	int j = threadIdx.y + blockDim.y*blockIdx.y;
	int i = threadIdx.z + blockDim.z*blockIdx.z;

	// Read the needed data from the information structure
	int rank = information_cuda->rank;
	int Nx = information_cuda->global_Nx;
	int Ny = information_cuda->global_Ny;
	int Nz = information_cuda->global_Nz;
	int loc_Nx = information_cuda->loc_Nx[rank];
	int loc_Ny = information_cuda->loc_Ny[rank];
	int loc_Nz = information_cuda->loc_Nz[rank];

    int I, J, K;
	I = loc_Nz; J = loc_Ny; K = loc_Nx;

	// Setting up steps
	double hi = 2.0/(Nz-1.0);
	double hj = 2.0/(Ny-1.0);
	double hk = 2.0/(Nx-1.0);
	double stepi = hi*hi;
	double stepj = hj*hj;
	double stepk = hk*hk;
	double f3 = 1.0/3.0;
	double f6 = 1.0/6.0;

	// Interior case
	// Consider optimizing this as kernels for i \in {0, 1, K-2, K-1} are not used
	if ( 
		( (i > 1) && (j > 0) && (k > 0)) 
		&& ((i < (I-2)) && (j < (J-1)) && (k < (K-1) )) ) 
	{
		// Compute new value
		// Save i, j, k index once
		int ijk = IND_3D(i, j, k, I, J, K);

		// Linear indexing with macro
		double ui = U_cuda[IND_3D(i - 1, j, k, I, J, K)] 
			+ U_cuda[IND_3D(i + 1, j, k, I, J, K)] 
			+ f3 * stepi * F_cuda[ijk];
		double uj = U_cuda[IND_3D(i, j - 1, k, I, J, K)] 
			+ U_cuda[IND_3D(i, j + 1, k, I, J, K)] 
			+ f3 * stepj * F_cuda[ijk];
		double uk = U_cuda[IND_3D(i, j, k - 1, I, J, K)] 
			+ U_cuda[IND_3D(i, j, k + 1, I, J, K)] 
			+ f3 * stepk * F_cuda[ijk];

		// Collect terms
		Unew_cuda[ijk] = f6 * (ui + uj + uk);
	}
}

// Kernel for boundary points. Starts being used in mixed_3
__global__ void jacobi_iteration_kernel_boundary(Information *information_cuda,
	double *U_cuda, double *F_cuda, double *Unew_cuda)
{
	// Read the needed data from the information structure
	int rank = information_cuda->rank;
	int Nx = information_cuda->global_Nx;
	int Ny = information_cuda->global_Ny;
	int Nz = information_cuda->global_Nz;
	int loc_Nx = information_cuda->loc_Nx[rank];
	int loc_Ny = information_cuda->loc_Ny[rank];
	int loc_Nz = information_cuda->loc_Nz[rank];

    int I, J, K;
	I = loc_Nz; J = loc_Ny; K = loc_Nx;

	// Setting up steps
	double hi = 2.0/(Nz-1.0);
	double hj = 2.0/(Ny-1.0);
	double hk = 2.0/(Nx-1.0);
	double stepi = hi*hi;
	double stepj = hj*hj;
	double stepk = hk*hk;
	double f3 = 1.0/3.0;
	double f6 = 1.0/6.0;

	
	// Determine where the thread is located
	int k = threadIdx.x + blockDim.x*blockIdx.x;
	int j = threadIdx.y + blockDim.y*blockIdx.y;
	int i = ( (threadIdx.z + blockDim.z*blockIdx.z) == 0) ? 1 : I - 2;

	// Boundary case
	if ( 
		( (j > 0) && (k > 0)) 
		&& ( (j < (J-1)) && (k < (K-1) )) ) 
	{
		// Compute new value
		// Do for first boundary (i = 1)
		int ijk = IND_3D(i, j, k, I, J, K);

		// Linear indexing with macro
		double ui = U_cuda[IND_3D(i - 1, j, k, I, J, K)] 
			+ U_cuda[IND_3D(i + 1, j, k, I, J, K)] 
			+ f3 * stepi * F_cuda[ijk];
		double uj = U_cuda[IND_3D(i, j - 1, k, I, J, K)] 
			+ U_cuda[IND_3D(i, j + 1, k, I, J, K)] 
			+ f3 * stepj * F_cuda[ijk];
		double uk = U_cuda[IND_3D(i, j, k - 1, I, J, K)] 
			+ U_cuda[IND_3D(i, j, k + 1, I, J, K)] 
			+ f3 * stepk * F_cuda[ijk];

		// Collect terms
		Unew_cuda[ijk] = f6 * (ui + uj + uk);
	}
}

// ============================================================================
// Function to compute the frobenious norm 

void compute_relative_norm_cuda(
	Information *information, Information *information_cuda,
	double *U_cuda, double *Unew_cuda)
{
	int N = 1;
	frobenious_kernel<<<1,N>>>(information_cuda,U_cuda,Unew_cuda);

	copy_from_device(
		&information->local_frobenius, sizeof(double),
		&information_cuda->local_frobenius
	);
}
__global__ void frobenious_kernel(
	Information *information_cuda, double *U_cuda, double *Unew_cuda)
{
	double local_frobenius=0.0;
	int rank = information_cuda->rank;
	int loc_Nx = information_cuda->loc_Nx[rank];
	int loc_Ny = information_cuda->loc_Ny[rank];
	int loc_Nz = information_cuda->loc_Nz[rank];
	
	int I, J, K;
	I = loc_Nz; J = loc_Ny; K = loc_Nx;

	//local_frobenius[threadIdx.x] = 0.0;
	// Loop over all interior points
	for (int ijk = threadIdx.x; ijk < I*J*K; ijk += blockDim.x) {
		//int ijk = IND_3D(i, j, k, I, J, K);
		double uij    = U_cuda[ijk];
		double unewij = Unew_cuda[ijk];
		local_frobenius += (uij - unewij)*(uij - unewij);
	}
	information_cuda->local_frobenius += local_frobenius;
}