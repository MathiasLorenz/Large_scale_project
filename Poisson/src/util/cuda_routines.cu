#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "matrix_routines.h"
#include "cuda_routines.h"
#include "jacobi_util.h"

// ============================================================================
// ALLOCATE AND DEALLOCATE DATA ON THE DEVICE

void cuda_malloc(void **device_array, int N_bytes)
{
	checkCudaErrors(hipMalloc(device_array , N_bytes));
}
void cuda_malloc_host(void **host_array, int N_bytes)
{
	checkCudaErrors(hipHostMalloc(host_array, N_bytes));
}

void cuda_free(double *device_array)
{
	checkCudaErrors(hipFree(device_array));
}
void cuda_host_free(double *host_array)
{
	checkCudaErrors(hipHostFree(host_array));
}

// ============================================================================
// COPY DATA TO AND FROM DEVICE

void copy_to_device(double *host, int N, double *device)
{
	checkCudaErrors(hipMemcpyAsync(device, host, N, hipMemcpyHostToDevice));
}
void copy_from_device(double *host, int N, double *device)
{
	checkCudaErrors(hipMemcpyAsync(host, device, N, hipMemcpyDeviceToHost));
}

// ============================================================================
// UTILITY FUNCTIONS

void cuda_synchronize(){
	checkCudaErrors(hipDeviceSynchronize());
}
void setCudaDevice(int rank)
{
	int Ndevices, device;
	hipGetDeviceCount( &Ndevices );
	if (Ndevices > 1)
		device = (rank % 2 == 0) ? 0 : 1;
	else 
		device = 0;
	hipSetDevice(device);
}