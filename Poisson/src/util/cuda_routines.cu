#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "matrix_routines.h"
#include "cuda_routines.h"
#include "jacobi_util.h"

// ============================================================================
// ALLOCATE AND DEALLOCATE DATA ON THE DEVICE

void cuda_malloc(void **device_array, int N_bytes)
{
	checkCudaErrors(hipMalloc(device_array , N_bytes));
}
void cuda_malloc_host(void **host_array, int N_bytes)
{
	checkCudaErrors(hipHostMalloc(host_array, N_bytes));
}

void cuda_free(double *device_array)
{
	checkCudaErrors(hipFree(device_array));
}
void cuda_host_free(double *host_array)
{
	checkCudaErrors(hipHostFree(host_array));
}

// ============================================================================
// COPY DATA TO AND FROM DEVICE

void copy_to_device_async(double *host, int N_bytes, double *device, void *stream)
{
	hipStream_t *s = (hipStream_t*)stream;
	checkCudaErrors(hipMemcpyAsync(device, host, N_bytes, hipMemcpyHostToDevice, *s));
}
void copy_from_device_async(double *host, int N_bytes, double *device, void *stream)
{
	hipStream_t *s = (hipStream_t*)stream;
	checkCudaErrors(hipMemcpyAsync(host, device, N_bytes, hipMemcpyDeviceToHost, *s));
}
void copy_to_device(double *host, int N_bytes, double *device)
{
	checkCudaErrors(hipMemcpy(device, host, N_bytes, hipMemcpyHostToDevice));
}
void copy_from_device(double *host, int N_bytes, double *device)
{
	checkCudaErrors(hipMemcpy(host, device, N_bytes, hipMemcpyDeviceToHost));
}
/*
void copy_from_device_void(void *host, int N_bytes, void *device)
{
	checkCudaErrors(hipMemcpy(host, device, N_bytes, hipMemcpyDeviceToHost));
}
*/

// ============================================================================
// UTILITY FUNCTIONS

// Synchronization
void cuda_synchronize(){
	checkCudaErrors(hipDeviceSynchronize());
}
void cuda_stream_synchronize(void *stream){
	hipStream_t *s = (hipStream_t*)stream;
	checkCudaErrors(hipStreamSynchronize(*s));
}

// Stream management
void cuda_create_stream(void **stream)
{
	*stream = malloc(sizeof(hipStream_t));
	hipStream_t streamT;
	hipStreamCreate(&streamT);
	memcpy(*stream,&streamT,sizeof(hipStream_t));
}
void cuda_destroy_stream(void *stream)
{
	hipStream_t *s = (hipStream_t*)stream;
	hipStreamDestroy(*s);
	free(stream);	
}

// Device management
void cuda_get_device_count(int *count)
{
	hipGetDeviceCount( count );
}
void cuda_set_device(int rank)
{
	int Ndevices, device;
	hipGetDeviceCount( &Ndevices );
	if (Ndevices > 1)
		device = (rank % 2 == 0) ? 0 : 1;
	else 
		device = 0;
	hipSetDevice(device);
}


// Enable peer access between GPU 0 and 1. Restores currently chosen GPU
void cuda_enable_peer_access()
{
	// Get the current device, s.t. it can be set afterwards.
	int current_device;
	hipGetDevice(&current_device);

	// Enable peer access
	hipSetDevice(0);
	checkCudaErrors(hipDeviceEnablePeerAccess(1, 0));
	hipSetDevice(1);
	checkCudaErrors(hipDeviceEnablePeerAccess(0, 0));

	// Set current device back
	hipSetDevice(current_device);
}