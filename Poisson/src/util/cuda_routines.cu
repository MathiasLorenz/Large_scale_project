#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "matrix_routines.h"
#include "cuda_routines.h"
#include "jacobi_util.h"

// ============================================================================
// ALLOCATE AND DEALLOCATE DATA ON THE DEVICE

void cuda_malloc(void** device_array, int N){
	checkCudaErrors(hipMalloc( device_array ,N));
}
void cuda_free(double *device_array){
	checkCudaErrors(hipFree(device_array));
}

// ============================================================================
// COPY DATA TO AND FROM DEVICE

void copy_to_device(double *host, int N, double *device){
	checkCudaErrors(hipMemcpyAsync(device, host, N, hipMemcpyHostToDevice));
}
void copy_from_device(double *host, int N, double *device){
	checkCudaErrors(hipMemcpyAsync(host, device, N, hipMemcpyDeviceToHost));
}

// ============================================================================
// UTILITY FUNCTIONS

void cuda_synchronize(){
	checkCudaErrors(hipDeviceSynchronize());
}