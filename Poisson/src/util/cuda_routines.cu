#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "matrix_routines.h"
#include "cuda_routines.h"
#include "jacobi_util.h"

// ============================================================================
// ALLOCATE AND DEALLOCATE DATA ON THE DEVICE

void cuda_malloc(void **device_array, int N_bytes)
{
	checkCudaErrors(hipMalloc(device_array , N_bytes));
}
void cuda_malloc_host(void **host_array, int N_bytes)
{
	checkCudaErrors(hipHostMalloc(host_array, N_bytes));
}

void cuda_free(double *device_array)
{
	checkCudaErrors(hipFree(device_array));
}
void cuda_host_free(double *host_array)
{
	checkCudaErrors(hipHostFree(host_array));
}

// ============================================================================
// COPY DATA TO AND FROM DEVICE

void copy_to_device_async(double *host, int N_bytes, double *device)
{
	checkCudaErrors(hipMemcpyAsync(device, host, N_bytes, hipMemcpyHostToDevice));
}
void copy_from_device_async(double *host, int N_bytes, double *device)
{
	checkCudaErrors(hipMemcpyAsync(host, device, N_bytes, hipMemcpyDeviceToHost));
}
void copy_to_device(double *host, int N_bytes, double *device)
{
	checkCudaErrors(hipMemcpy(device, host, N_bytes, hipMemcpyHostToDevice));
}
void copy_from_device(double *host, int N_bytes, double *device)
{
	checkCudaErrors(hipMemcpy(host, device, N_bytes, hipMemcpyDeviceToHost));
}
/*
void copy_from_device_void(void *host, int N_bytes, void *device)
{
	checkCudaErrors(hipMemcpy(host, device, N_bytes, hipMemcpyDeviceToHost));
}
*/

// ============================================================================
// UTILITY FUNCTIONS

void cuda_synchronize(){
	checkCudaErrors(hipDeviceSynchronize());
}
void cuda_get_device_count(int *count)
{
	hipGetDeviceCount( count );
}
void cuda_set_device(int rank)
{
	int Ndevices, device;
	hipGetDeviceCount( &Ndevices );
	if (Ndevices > 1)
		device = (rank % 2 == 0) ? 0 : 1;
	else 
		device = 0;
	hipSetDevice(device);
}


// Enable peer access
void cuda_enable_peer_access(const int access_from, const int access_to)
{
	hipSetDevice(access_from);
	checkCudaErrors(hipDeviceEnablePeerAccess(access_to, 0));
}