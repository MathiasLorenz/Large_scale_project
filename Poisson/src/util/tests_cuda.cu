// ============================================================================
// INCLUDES
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "tests_cuda.h"
#include "cuda_routines.h"

// ============================================================================
// CUDA TEST
void test_cuda(int Nx, int Ny, int Nz)
{
	printf("We are now testing the cuda function\n");
	cu_test<<<1,1>>>();
	checkCudaErrors(hipDeviceSynchronize());
}

